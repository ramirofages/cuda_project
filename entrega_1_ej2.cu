#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
#include <math.h>

#include <sys/time.h>
#include <sys/resource.h>


double dwalltime(){
        double sec;
        struct timeval tv;

        gettimeofday(&tv,NULL);
        sec = tv.tv_sec + tv.tv_usec/1000000.0;
        return sec;
}



double* arreglo;
double* suma_total;

double* d_arreglo_suma;
double* d_arreglo;
double* d_arreglo_2;

int cant_elem = 2048000;

void init_CPU_array_float(double* v, int n){
	for(int i = 0; i < n; i++) {
		v[i] = (double)i;
	}
}

// realiza la suma total de forma paralela, aumentando el offset en cada ejecucion
__global__ void sumador(double* arreglo, int offset, int N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid < N){

		if( (tid & ( (offset * 2) -1)) == 0 && ( (tid+offset) < N))
		{

			arreglo[tid] = arreglo[tid] + arreglo[tid + offset];

		}


	}

}

// (V[i] +/- promedio)^2 --- multiplicador es 1 o -1 dependiendo de si se quiere sumar o restar
__global__ void suma_prom(double* arreglo, int multiplicador, double promedio, int N){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N)
	{
		double num = arreglo[tid] + (multiplicador * promedio);
		arreglo[tid] = num * num;
	}
}

double solucion_CPU(){

	double prom = 0;

	// sumamos todos los elementos para calcular el promedio
	for (int i = 0; i < cant_elem; ++i)
	{
		prom = prom + arreglo[i];
	}
	prom /= cant_elem;

	double dividendo = 0;
	double divisor = 0;

	// realizamos la sumatoria del dividendo y divisor
	for (int i = 0; i < cant_elem; ++i)
	{
		double num = arreglo[i] - prom;
		double num2 = arreglo[i] + prom;

		dividendo +=  (num*num);
		divisor += (num2*num2);
	}

	divisor = divisor + 1;

	return sqrt(dividendo/divisor);
}


int main(int argc, char** argv){



	double timetick;
	int numBytes = sizeof(double) * cant_elem; //bytes a alocar

	arreglo = (double*) malloc(numBytes);
	suma_total = (double *) malloc(sizeof(double));  // usado para traer el primer elemento resultado de la suma paralela

	init_CPU_array_float(arreglo, cant_elem);



	hipMalloc(&d_arreglo, numBytes);
	hipMalloc(&d_arreglo_2, numBytes);
	hipMalloc(&d_arreglo_suma, numBytes);


	hipMemcpy(d_arreglo, arreglo, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_arreglo_2, arreglo, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_arreglo_suma, arreglo, numBytes, hipMemcpyHostToDevice);

	dim3 miGrid1D(4000,1);
	dim3 miBloque1D(512,1);




	timetick = dwalltime();

	hipError_t error;
	// Sumamos todos los elementos para el promedio, el resultado queda almacenado en la primer posicion
	for(int i=1; i < cant_elem; i*= 2){
		sumador<<<miGrid1D, miBloque1D>>>(d_arreglo_suma, i, cant_elem);
		hipDeviceSynchronize();

	}



	// Esperamos a que termine la ejecucion
 	
 	//printf("%d\n", error);

 	// Traemos el primer elemento de d_arreglo_suma el cual posee el resultado
	hipMemcpy(suma_total, d_arreglo_suma, sizeof(double), hipMemcpyDeviceToHost);


	double promedio = (*suma_total) / cant_elem;


	// ############################################
	// ############################################
	// Dividendo
	suma_prom<<<miGrid1D, miBloque1D>>>(d_arreglo_2, -1, promedio, cant_elem);

	for(int i=1; i < cant_elem; i*=2){
		sumador<<<miGrid1D, miBloque1D>>>(d_arreglo_2, i, cant_elem);
	}

	error = hipDeviceSynchronize();
 	//printf("%d\n", error);
	hipMemcpy(suma_total, d_arreglo_2, sizeof(double), hipMemcpyDeviceToHost);

	double dividendo = (*suma_total);

	// ############################################
	// ############################################
	// Divisor
	suma_prom<<<miGrid1D, miBloque1D>>>(d_arreglo, 1, promedio, cant_elem);

	for(int i=1; i < cant_elem; i*=2){
		sumador<<<miGrid1D, miBloque1D>>>(d_arreglo, i, cant_elem);
	}

	error = hipDeviceSynchronize();
 	//printf("%d\n", error);
	printf("-> Tiempo transcurrido en la GPU %f\n", dwalltime() - timetick);

	hipMemcpy(suma_total, d_arreglo, sizeof(double), hipMemcpyDeviceToHost);

	double divisor = *suma_total + 1;
	// ############################################

	double division = dividendo / divisor;
	double resultado = sqrt(division);

	printf("Resultado GPU: %f\n", resultado);

	timetick = dwalltime();

	double cpu_result = solucion_CPU();

	printf("-> Tiempo transcurrido en la CPU %f\n", dwalltime() - timetick);
	printf("Resultado CPU: %f\n", cpu_result);


	free(arreglo);
	free(suma_total);
	hipFree (d_arreglo);
	hipFree (d_arreglo_2);
	hipFree (d_arreglo_suma);

}
