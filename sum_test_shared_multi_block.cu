#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

#define threads_per_block 10



void printi(int i){
	printf("%d\n", i);
}


void init_CPU_array(int* array, int n){
	for(int i = 0; i < n; i++) {
		array[i] = 1;
	}
}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}


// realiza la suma de determinantes
__global__ void sumador(int* arreglo, int* result, float N)
{
	__shared__ int compartida[threads_per_block];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	compartida[threadIdx.x] = arreglo[tid];
	__syncthreads();
	for(int i=1; pow((float)2,(float)i-1) < 10; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1);
		if(threadIdx.x < (10.0/acceso))
		{
				if((threadIdx.x * acceso + offset) < (N - blockIdx.x * blockDim.x))
				{
					compartida[threadIdx.x * acceso] = compartida[threadIdx.x * acceso] + compartida[threadIdx.x * acceso + offset];
					compartida[threadIdx.x * acceso + offset] = 0;
				}
				
				// printf("%s %d\n", "TRABAJA id:", threadIdx.x);
				// printf("%s %d\n", "OFFSET:", threadIdx.x * acceso + offset);
				// printf("%s %d\n", "result:", compartida[threadIdx.x * acceso]);
				result[blockIdx.x] = compartida[0];

		}

		// printf("%s\n", "");

		
	}

}



int* arreglo_suma1;
int* d_arreglo_suma1;

int* arreglo_result;
int* d_arreglo_suma2;

int main(int argc, char** argv){

	int N = 110;
	//##################################################################################
	//############################## INICIALIZACION ####################################

	arreglo_suma1 = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_suma1, N * sizeof(int));

	arreglo_result = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_suma2, N * sizeof(int));


	init_CPU_array(arreglo_suma1, N);
	hipMemcpy(d_arreglo_suma1, arreglo_suma1, N * sizeof(int), hipMemcpyHostToDevice);

	//float threads_per_block = 10;
	int block_count = ceil((float)N / threads_per_block);
	printf("block count %d\n", block_count);

	//##################################################################################
	//################################ EJECUCIONES #####################################

	dim3 miGrid1D_1(block_count,1);
	dim3 miBloque1D_1(threads_per_block,1);
	sumador<<<miGrid1D_1, miBloque1D_1>>>(d_arreglo_suma1, d_arreglo_suma2, N);
 	hipDeviceSynchronize();

	int remaining_elements = ceil((float)N/threads_per_block);
	printf("fin 1, elementos restantes: %d\n", remaining_elements);

	dim3 miGrid1D_2(2,1);
	dim3 miBloque1D_2(threads_per_block,1);
	sumador<<<miGrid1D_2, miBloque1D_2>>>(d_arreglo_suma2, d_arreglo_suma1, 11);
 	hipDeviceSynchronize();

	// remaining_elements = ceil((float)N/threads_per_block/threads_per_block);
	// printf("fin 2, elementos restantes: %d\n", remaining_elements);

	dim3 miGrid1D_3(1,1);
	dim3 miBloque1D_3(threads_per_block,1);
	sumador<<<miGrid1D_3, miBloque1D_3>>>(d_arreglo_suma1, d_arreglo_suma2, 2);
 	hipDeviceSynchronize();

	// remaining_elements = ceil((float)N/threads_per_block/threads_per_block/threads_per_block);
	// printf("fin 3, elementos restantes: %d\n", remaining_elements);

	//##################################################################################
	//################################### READ BACK #####################################

	hipMemcpy(arreglo_result, d_arreglo_suma2, N * sizeof(int), hipMemcpyDeviceToHost);

	printf("%s\n", "RESULTADO DE LA SUMA:");
	print_CPU_array(arreglo_result, 15);

	free(arreglo_suma1);
	hipFree (d_arreglo_suma1);

	free(arreglo_result);
	hipFree (d_arreglo_suma2);

}