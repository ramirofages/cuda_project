#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

#define threads_per_block 5



void printi(int i){
	printf("%d\n", i);
}


void init_CPU_array(int* array, int n){
	for(int i = 0; i < n; i++) {
		array[i] = 1;
	}
}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}


// realiza la suma de determinantes
__global__ void sumador(int* arreglo, int* result, float N)
{
	__shared__ int compartida[threads_per_block];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	compartida[threadIdx.x] = arreglo[tid];
	__syncthreads();
	for(int i=1; pow((float)2,(float)i-1) < N; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1);
		if(threadIdx.x < (N/acceso))
		{
				compartida[threadIdx.x * acceso] = compartida[threadIdx.x * acceso] + compartida[threadIdx.x * acceso + offset];
				compartida[threadIdx.x * acceso + offset] = 0;
				// printf("%s %d\n", "TRABAJA id:", threadIdx.x);
				result[blockIdx.x] = compartida[0];

		}

		// printf("%s\n", "");

		
	}

}





int* arreglo_suma1;
int* d_arreglo_suma1;

int* arreglo_result;
int* d_arreglo_suma2;

int main(int argc, char** argv){

	int N = 26;
	//##################################################################################
	//############################## INICIALIZACION ####################################

	arreglo_suma1 = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_suma1, N * sizeof(int));

	arreglo_result = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_suma2, N * sizeof(int));


	init_CPU_array(arreglo_suma1, N);
	hipMemcpy(d_arreglo_suma1, arreglo_suma1, N * sizeof(int), hipMemcpyHostToDevice);

	//float threads_per_block = 10;
	int block_count = ceil((float)N / threads_per_block);

	//##################################################################################
	//################################ EJECUCIONES #####################################

	dim3 miGrid1D_1(block_count,1);
	dim3 miBloque1D_1(threads_per_block,1);
	sumador<<<miGrid1D_1, miBloque1D_1>>>(d_arreglo_suma1, d_arreglo_suma2, threads_per_block);
 	hipDeviceSynchronize();

	printf("%s\n", "fin 1");

	dim3 miGrid1D_2(2,1);
	dim3 miBloque1D_2(threads_per_block,1);
	sumador<<<miGrid1D_2, miBloque1D_2>>>(d_arreglo_suma2, d_arreglo_suma1, threads_per_block);
 	hipDeviceSynchronize();

	printf("%s\n", "fin 2");

	//##################################################################################
	//################################### READ BACK #####################################

	hipMemcpy(arreglo_result, d_arreglo_suma1, N * sizeof(int), hipMemcpyDeviceToHost);

	printf("%s\n", "RESULTADO DE LA SUMA:");
	print_CPU_array(arreglo_result, N);

	free(arreglo_suma1);
	hipFree (d_arreglo_suma1);

	free(arreglo_result);
	hipFree (d_arreglo_suma2);

}