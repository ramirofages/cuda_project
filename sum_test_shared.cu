#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"


int N = 10;


void printi(int i){
	printf("%d\n", i);
}


void init_CPU_array(int* array, int n){
	for(int i = 0; i < n; i++) {
		array[i] = i;
	}
}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}


// realiza la suma de determinantes
__global__ void sumador_3(int* arreglo, int* result, float N){
	__shared__ int compartida[10];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	compartida[threadIdx.x] = arreglo[tid];
	__syncthreads();
	printf("%d\n", compartida[threadIdx.x]);
	for(int i=1; pow((float)2,(float)i-1) < N; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1);
		if(threadIdx.x < (N/acceso))
		{
				arreglo[threadIdx.x * acceso] = arreglo[threadIdx.x * acceso] + arreglo[threadIdx.x * acceso + offset];
				arreglo[threadIdx.x * acceso + offset] = 0;
				printf("%s\n", "TRABAJO");

		}
		printf("%s\n", "");
		
	}

}




int* arreglo_determinantes;
int* d_arreglo_determinantes;

int* arreglo_suma1;
int* d_arreglo_suma1;

int* arreglo_suma2;
int* d_arreglo_suma2;

int main(int argc, char** argv){

	int* suma_det = (int *) malloc(sizeof(int)); 

	arreglo_determinantes = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_determinantes, N * sizeof(int));

	arreglo_suma1 = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_suma1, N * sizeof(int));

	arreglo_suma2 = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_suma2, N * sizeof(int));


	init_CPU_array(arreglo_determinantes, N);
	hipMemcpy(d_arreglo_determinantes, arreglo_determinantes, N * sizeof(int), hipMemcpyHostToDevice);

	dim3 miGrid1D_2(1,1);
	dim3 miBloque1D_2(N,1);

	sumador_3<<<miGrid1D_2, miBloque1D_2>>>(d_arreglo_determinantes, N);
	// for(int i=1; pow(2,i-1) < N; i++)
	// {
	// 	sumador_3<<<miGrid1D_2, miBloque1D_2>>>(d_arreglo_determinantes, (int)pow(2,i), (int)pow(2, i-1), N);
	// 	hipDeviceSynchronize();

	// 	printf("%s\n", " ");

	// }

	hipMemcpy(arreglo_determinantes, d_arreglo_determinantes, 10 * sizeof(int), hipMemcpyDeviceToHost);

	printf("%s\n", "TEST SUMA:");
	print_CPU_array(arreglo_determinantes, 10);

	free(arreglo_determinantes);
	hipFree (d_arreglo_determinantes);

}