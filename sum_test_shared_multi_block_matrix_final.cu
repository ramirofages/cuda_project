#include "hip/hip_runtime.h"
#include "stdio.h"
#define threads_per_block 10




void printi(int i){
	printf("%d\n", i);
}


void init_CPU_array(int* arreglo_b, int n){
	for(int i=0; i< n; i++)
	{
		arreglo_b[(i*16) + 0] = 1;
		arreglo_b[(i*16) + 1] = 1;
		arreglo_b[(i*16) + 2] = 1;
		arreglo_b[(i*16) + 3] = 1;
		arreglo_b[(i*16) + 4] = 1;
		arreglo_b[(i*16) + 5] = 1;
		arreglo_b[(i*16) + 6] = 1;
		arreglo_b[(i*16) + 7] = 1;
		arreglo_b[(i*16) + 8] = 1;
		arreglo_b[(i*16) + 9] = 1;
		arreglo_b[(i*16) + 10] = 1;
		arreglo_b[(i*16) + 11] = 1;
		arreglo_b[(i*16) + 12] = 1;
		arreglo_b[(i*16) + 13] = 1;
		arreglo_b[(i*16) + 14] = 1;
		arreglo_b[(i*16) + 15] = 1;

	}


}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}
void print_CPU_matrix(int array[], int n){
	for(int i = 0; i < n; i++) {
		if(i % 16 == 0)
			printf("%s\n", "");

		printf("%d ", array[i]);
	}
}


//realiza la suma de n matrices
__global__ void sumador_2(int* arreglo, int* result, float N){

	__shared__ int compartido[threads_per_block * 16];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	compartido[threadIdx.x] = arreglo[tid];
	__syncthreads();


	for(int i=1; pow((float)2,(float)i-1) < N; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1) ;
		if( (( (threadIdx.x / 16) & (offset*2)-1 ) == 0) && ( (threadIdx.x+offset*16) < N*16))
		{
				compartido[threadIdx.x] = compartido[threadIdx.x] + compartido[threadIdx.x + offset * 16];
		}
	}
	__syncthreads();

	if(threadIdx.x < 16)
	{
		result[blockIdx.x * 16 + threadIdx.x] = compartido[threadIdx.x];
	}
}




int* arreglo_suma1;
int* d_arreglo_suma1;

int* arreglo_result;
int* d_arreglo_suma2;

int main(int argc, char** argv){

	int N = 12;
	//##################################################################################
	//############################## INICIALIZACION ####################################
	int byte_size = N * sizeof(int) * 16;
	arreglo_suma1 = (int*) malloc(byte_size);
	hipMalloc(&d_arreglo_suma1, byte_size);

	arreglo_result = (int*) malloc(byte_size);
	hipMalloc(&d_arreglo_suma2, byte_size);


	init_CPU_array(arreglo_suma1, N);
	hipMemcpy(d_arreglo_suma1, arreglo_suma1, byte_size, hipMemcpyHostToDevice);


	//##################################################################################
	//################################ EJECUCIONES #####################################

	int block_count = ceil((float)(N) / (threads_per_block));
	// printf("block count %d\n", block_count);
	// dim3 miBloque1D_1(threads_per_block *16,1);
	// for(int i=1; pow(threads_per_block, i-1) < N; i++)
	// {
	// 	int remaining_elements = ceil((float)N/pow(threads_per_block, i-1));
	// 	int block_count = ceil((float)N/pow(threads_per_block * 16, i-1));

	// 	dim3 miGrid1D_1(block_count,1);
	// 	sumador_2<<<miGrid1D_1, miBloque1D_1>>>(d_arreglo_suma1, d_arreglo_suma2, N);
	// 	cudaThreadSynchronize();

	// 	int* tmp = d_arreglo_suma1;
	// 	d_arreglo_suma1 = d_arreglo_suma2;
	// 	d_arreglo_suma2 = tmp;

	// 	printf("elementos restantes: %d \n ", remaining_elements);
	// 	printf("block_count: %d \n ", block_count);


	// }


	dim3 miBloque1D_1(threads_per_block * 16,1);
	dim3 miGrid1D_1(block_count,1);
	sumador_2<<<miGrid1D_1, miBloque1D_1>>>(d_arreglo_suma1, d_arreglo_suma2, N);

	dim3 miGrid1D_2(1,1);
	sumador_2<<<miGrid1D_2, miBloque1D_1>>>(d_arreglo_suma2, d_arreglo_suma1, N);

	// dim3 miGrid1D_3(1,1);
	// sumador_2<<<miGrid1D_3, miBloque1D_1>>>(d_arreglo_suma1, d_arreglo_suma2, N);

	//##################################################################################
	//################################### READ BACK #####################################

	hipMemcpy(arreglo_result, d_arreglo_suma1, N * sizeof(int) * 16, hipMemcpyDeviceToHost);

	printf("%s\n", "RESULTADO DE LA SUMA:");
	print_CPU_matrix(arreglo_result, 16 * 2);

	free(arreglo_suma1);
	hipFree (d_arreglo_suma1);

	free(arreglo_result);
	hipFree (d_arreglo_suma2);

}