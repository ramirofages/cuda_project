#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"


int N = 10;


void printi(int i){
	printf("%d\n", i);
}


void init_CPU_array(int* array, int n){
	for(int i = 0; i < n; i++) {
		array[i] = i;
	}
}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}


// realiza la suma de determinantes
__global__ void sumador_3(int* arreglo, int acceso, int offset, int i, float N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// if(tid < N/acceso)
	// {
	// 	printf("%d\n", arreglo[tid * acceso]);
	// 	printf("%d\n", arreglo[tid * acceso + offset]);
	// }
	if(tid < (N/acceso))
	{
			arreglo[tid * acceso] = arreglo[tid * acceso] + arreglo[tid * acceso + offset];
			arreglo[tid * acceso + offset] = 0;
			printf("%s\n", "TRABAJO");

	}
}




int* arreglo_determinantes;
int* d_arreglo_determinantes;

int main(int argc, char** argv){

	int* suma_det = (int *) malloc(sizeof(int)); 

	arreglo_determinantes = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_determinantes, N * sizeof(int));

	init_CPU_array(arreglo_determinantes, N);
	hipMemcpy(d_arreglo_determinantes, arreglo_determinantes, N * sizeof(int), hipMemcpyHostToDevice);

	dim3 miGrid1D_2(1,1);
	dim3 miBloque1D_2(N,1);

	for(int i=1; i < N; i++)
	{
		sumador_3<<<miGrid1D_2, miBloque1D_2>>>(d_arreglo_determinantes, (int)pow(2,i), (int)pow(2, i-1), i, N);
		hipDeviceSynchronize();
		
		printf("%s\n", "Acceso:");
		printf("%d\n", (int)pow(2,i));

		printf("%s\n", "Offset:");
		printf("%d\n", (int)pow(2,i-1));

		printf("%s\n", " ");


		if(i==4) break;
	}

	hipMemcpy(arreglo_determinantes, d_arreglo_determinantes, 10 * sizeof(int), hipMemcpyDeviceToHost);

	printf("%s\n", "TEST SUMA:");
	//printf("%d\n", *suma_det);
	print_CPU_array(arreglo_determinantes, 10);

	free(arreglo_determinantes);
	hipFree (d_arreglo_determinantes);

}