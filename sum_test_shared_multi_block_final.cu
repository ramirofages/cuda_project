#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"




void printi(int i){
	printf("%d\n", i);
}


void init_CPU_array(int* array, int n){
	for(int i = 0; i < n; i++) {
		array[i] = i;
	}
}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}


// realiza la suma de determinantes
__global__ void sumador(int* arreglo, int* result, float N)
{
	__shared__ int compartida[10];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	compartida[threadIdx.x] = arreglo[tid];
	__syncthreads();
	for(int i=1; pow((float)2,(float)i-1) < N; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1);
		if(threadIdx.x < (N/acceso))
		{
				compartida[threadIdx.x * acceso] = compartida[threadIdx.x * acceso] + compartida[threadIdx.x * acceso + offset];
				compartida[threadIdx.x * acceso + offset] = 0;
				// printf("%s %d\n", "TRABAJA id:", threadIdx.x);
				result[blockIdx.x] = compartida[0];

		}

		// printf("%s\n", "");

		
	}

}





int* arreglo_suma1;
int* d_arreglo_suma1;

int* arreglo_result;
int* d_arreglo_suma2;

int main(int argc, char** argv){

	int N = 20;
	//##################################################################################
	//############################## INICIALIZACION ####################################

	arreglo_suma1 = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_suma1, N * sizeof(int));

	arreglo_result = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_suma2, N * sizeof(int));


	init_CPU_array(arreglo_suma1, N);
	hipMemcpy(d_arreglo_suma1, arreglo_suma1, N * sizeof(int), hipMemcpyHostToDevice);

	int block_count = 2;
	int threads_per_block = N/block_count;

	//##################################################################################
	//################################ EJECUCIONES #####################################

	dim3 miGrid1D_1(block_count,1);
	dim3 miBloque1D_1(threads_per_block,1);
	sumador<<<miGrid1D_1, miBloque1D_1>>>(d_arreglo_suma1, d_arreglo_suma2, 10);
 	hipDeviceSynchronize();

	printf("%s\n", "");
	printf("%s\n", "fin 1");

	dim3 miGrid1D_2(block_count,1);
	dim3 miBloque1D_2(threads_per_block,1);
	sumador<<<miGrid1D_2, miBloque1D_2>>>(d_arreglo_suma2, d_arreglo_suma1, 10);
 	hipDeviceSynchronize();

	printf("%s\n", "");
	printf("%s\n", "fin 2");


	for(int i=1; i < 220; i*=10)
	{
		int block_amount = ceil(220.0 / (i*10.0));
		// sumador<<<miGrid1D_1, miBloque1D_1>>>(d_arreglo_suma1, d_arreglo_suma2, 10);
		// hipDeviceSynchronize();
		printf("%s\n", " ");
		printf("index: %d \n ", block_amount);


	}

	//##################################################################################
	//################################### READ BACK #####################################

	hipMemcpy(arreglo_result, d_arreglo_suma1, N * sizeof(int), hipMemcpyDeviceToHost);

	printf("%s\n", "RESULTADO DE LA SUMA:");
	//print_CPU_array(arreglo_result, N);

	free(arreglo_suma1);
	hipFree (d_arreglo_suma1);

	free(arreglo_result);
	hipFree (d_arreglo_suma2);

}