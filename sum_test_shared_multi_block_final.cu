#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
#define threads_per_block 512




void printi(int i){
	printf("%d\n", i);
}


void init_CPU_array(int* array, int n){
	for(int i = 0; i < n; i++) {
		array[i] = 1;
	}
}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}


// realiza la suma de determinantes
__global__ void sumador(int* arreglo, int* result, float N)
{
	__shared__ int compartida[threads_per_block];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid > N)
	{
		return;
	}

	compartida[threadIdx.x] = arreglo[tid];
		__syncthreads();
		for(int i=1; pow((float)2,(float)i-1) < threads_per_block; i++)
		{
			int acceso = pow((float)2,(float)i);
			int offset = pow((float)2, (float)i-1);
			if(threadIdx.x < ((float)threads_per_block/acceso) && (threadIdx.x * acceso + offset) < (N - blockIdx.x * blockDim.x))
			{
					compartida[threadIdx.x * acceso] = compartida[threadIdx.x * acceso] + compartida[threadIdx.x * acceso + offset];
					// compartida[threadIdx.x * acceso + offset] = 0;
			}
			__syncthreads();

		}

		//el primer thread de cada grupo guarda el resultado

		if(threadIdx.x == 0)
			result[blockIdx.x] = compartida[0];
	


}





int* arreglo_suma1;
int* d_arreglo_suma1;

int* arreglo_result;
int* d_arreglo_suma2;

int main(int argc, char** argv){

	int N = 1024000;
	//##################################################################################
	//############################## INICIALIZACION ####################################

	arreglo_suma1 = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_suma1, N * sizeof(int));

	arreglo_result = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_suma2, N * sizeof(int));


	init_CPU_array(arreglo_suma1, N);
	hipMemcpy(d_arreglo_suma1, arreglo_suma1, N * sizeof(int), hipMemcpyHostToDevice);

	// int threads_per_block = 10;
	// int block_count = ceil((float)N / threads_per_block);

	//##################################################################################
	//################################ EJECUCIONES #####################################

	dim3 miBloque1D_1(threads_per_block,1);
	for(int i=0; pow(threads_per_block, i) < N ; i++)
	{
		int remaining_elements = ceil((float)N/pow(threads_per_block, i));
		int block_count = ceil((float)N/pow(threads_per_block, i+1));
		dim3 miGrid1D_1(block_count,1);
		sumador<<<miGrid1D_1, miBloque1D_1>>>(d_arreglo_suma1, d_arreglo_suma2, remaining_elements);
		hipDeviceSynchronize();

		int* tmp = d_arreglo_suma1;
		d_arreglo_suma1 = d_arreglo_suma2;
		d_arreglo_suma2 = tmp;

		printf("elementos restantes: %d \n", remaining_elements);
		printf("bloques usados:      %d \n\n", block_count);


	}

	//##################################################################################
	//################################### READ BACK #####################################

	hipMemcpy(arreglo_result, d_arreglo_suma1, N * sizeof(int), hipMemcpyDeviceToHost);

	printf("%s\n", "RESULTADO DE LA SUMA:");
	print_CPU_array(arreglo_result, 1);

	free(arreglo_suma1);
	hipFree (d_arreglo_suma1);

	free(arreglo_result);
	hipFree (d_arreglo_suma2);

}