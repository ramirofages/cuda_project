#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"


#include <sys/time.h>
#include <sys/resource.h>


double dwalltime(){
        double sec;
        struct timeval tv;

        gettimeofday(&tv,NULL);
        sec = tv.tv_sec + tv.tv_usec/1000000.0;
        return sec;
}


int cant = 512;
int cant_elem = cant * cant;

// arreglos usados como matrices
int* arreglo_A;
int* arreglo_B;
int* arreglo_C;


int* d_arreglo_A;
int* d_arreglo_B;
int* d_arreglo_C;

int* d_arreglo_AT;
int* d_arreglo_BT;


void printi(int i){
	printf("%d\n", i);
}

void init_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		array[i] = i;
	}
}

void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}

// calcula la transpuesta out-place
__global__ void transposeador(int* arreglo_b, int* arreglo_bt, int N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int i = (int)tid / N; // columna
	int j = (int)tid % N; // fila
	if((i<N) && (j<N)){
		arreglo_bt[i*N+j] = arreglo_b[j*N+i];
	}

}


// copia B en C
__global__ void copiador(int* arreglo_b, int* arreglo_c, int N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N)
		arreglo_c[tid] = arreglo_b[tid];
}

// C += A
__global__ void sumador(int* arreglo_a, int* arreglo_c, int N){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N)
		arreglo_c[tid] += arreglo_a[tid];
}

// C += A * B^t
__global__ void multiplicador(int* arreglo_a, int* arreglo_b_trans, int* arreglo_c, int N, int total_elem){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int i = (int)tid / N; // columna
	int j = (int)tid % N; // fila

	int k;
	int cuenta = 0;
	if(tid < total_elem)
	{
		for (k=0; k< N; k++){
			cuenta += arreglo_a[i*N+k] * arreglo_b_trans[k*N+j];
		}
		arreglo_c[tid] += cuenta;
	}

}


void solucion_CPU(){
	int* arreglo_at;
	int* arreglo_bt;
	int* arreglo_a_por_b;
	int* arreglo_res;


	int numBytes = sizeof(int) * cant_elem; //bytes a alocar

	arreglo_at = (int *) malloc(numBytes);
	arreglo_bt = (int *) malloc(numBytes);
	arreglo_a_por_b = (int *) malloc(numBytes); // resultado de A * B^t
	arreglo_res = (int *) malloc(numBytes);

	double timetick;
	timetick = dwalltime();

	// calculamos la transpeusta de arreglo_B
	for (int i = 0; i < cant_elem; ++i)
	{
		int col = i / cant; // columna
		int row = i % cant; // fila
		arreglo_bt[col*cant+row] = arreglo_B[row*cant+col];
	}

	// calculamos la transpeusta de arreglo_A
	for (int i = 0; i < cant_elem; ++i)
	{
		int col = i / cant; // columna
		int row = i % cant; // fila
		arreglo_at[col*cant+row] = arreglo_A[row*cant+col];
	}

	// A * B^t
	for (int i = 0; i < cant_elem; i++)
	{
		int col = i / cant; // columna
		int row = i % cant; // fila
		int mul = 0;
		for (int k=0; k< cant; k++){
			mul += arreglo_A[col*cant+k] * arreglo_bt[k*cant+row];
		}
		arreglo_a_por_b[i] = mul;
	}

	for (int i = 0; i < cant_elem; i++){
		arreglo_res[i] = 0;
	}
	// C = B + A * B^t + A^t
	for (int i = 0; i < cant_elem; i++){
		arreglo_res[i] += arreglo_B[i] + arreglo_a_por_b[i] + arreglo_at[i];
	}

	printf("-> Tiempo transcurrido en la CPU %f\n", dwalltime() - timetick);
	
	// printf("%s\n", "");
	// printf("%s\n", "Resultados CPU:");
	// for (int i = 0; i < cant_elem; i++){
	// 	printf("%d\n", arreglo_res[i]);
	// }

	


	free(arreglo_at);
	free(arreglo_bt);
	free(arreglo_a_por_b);
	free(arreglo_res);
}


int main(int argc, char** argv){
	int numBytes = sizeof(int) * cant_elem; //bytes a alocar

	arreglo_A = (int *) malloc(numBytes);
	arreglo_B = (int *) malloc(numBytes);
	arreglo_C = (int *) malloc(numBytes);
	double timetick;

	// llenamos los arreglos
	init_CPU_array(arreglo_A, cant_elem);
	init_CPU_array(arreglo_B, cant_elem);
	init_CPU_array(arreglo_C, cant_elem);

	// allocamos memoria en la gpu
	hipMalloc(&d_arreglo_A, numBytes);
	hipMalloc(&d_arreglo_B, numBytes);
	hipMalloc(&d_arreglo_C, numBytes);

	hipMalloc(&d_arreglo_AT, numBytes);
	hipMalloc(&d_arreglo_BT, numBytes);

	// copiamos los datos de la cpu a la gpu
	hipMemcpy(d_arreglo_A, arreglo_A, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_arreglo_B, arreglo_B, numBytes, hipMemcpyHostToDevice);

	dim3 miGrid1D(512,1);
	dim3 miBloque1D(512,1);

	timetick = dwalltime();


	// C = B
	copiador<<<miGrid1D, miBloque1D>>>(d_arreglo_B, d_arreglo_C, cant_elem);

	// B^t
	transposeador<<<miGrid1D, miBloque1D>>>(d_arreglo_B, d_arreglo_BT, cant);


	// C += A * B^t
	multiplicador <<<miGrid1D, miBloque1D>>>(d_arreglo_A, d_arreglo_BT, d_arreglo_C, cant, cant_elem);

	// A^t
	transposeador<<<miGrid1D, miBloque1D>>>(d_arreglo_A, d_arreglo_AT, cant);

	// C += A^t
	sumador<<<miGrid1D, miBloque1D>>>(d_arreglo_AT, d_arreglo_C, cant_elem);

	// Esperamos a que termine la ejecucion
 	hipDeviceSynchronize();

	printf("-> Tiempo transcurrido en la GPU %f\n", dwalltime() - timetick);

	// nos traemos los resultados de la gpu a la cpu
	hipMemcpy(arreglo_C, d_arreglo_C, numBytes, hipMemcpyDeviceToHost);

	//imprimimos los resultados
	// printf("%s\n", "");
	// printf("%s\n", "Resultados GPU:");
	// print_CPU_array(arreglo_C, cant_elem);

	solucion_CPU();
	
	free(arreglo_A);
	free(arreglo_B);
	free(arreglo_C);
	hipFree (d_arreglo_A);
	hipFree (d_arreglo_B);
	hipFree (d_arreglo_C);

}
