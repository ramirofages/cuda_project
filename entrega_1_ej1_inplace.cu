#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

#include <sys/time.h>
#include <sys/resource.h>


double dwalltime(){
        double sec;
        struct timeval tv;

        gettimeofday(&tv,NULL);
        sec = tv.tv_sec + tv.tv_usec/1000000.0;
        return sec;
}

int cant = 512;
int cant_elem = cant * cant;



// arreglos usados como matrices
int* arreglo_A;
int* arreglo_B;
int* arreglo_C;

int* d_arreglo_A;
int* d_arreglo_B;
int* d_arreglo_C;


void printi(int i){
	printf("%d\n", i);
}

void init_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		array[i] = i;
	}
}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}

// calcula la transpuesta in-place
__global__ void transposeador(int* arreglo_b, int N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int i = int((1+sqrtf(1+8*tid))/2);
	int j = tid - (i*(i-1)/2);
	int aux;
	if((i<N) && (j<N)){
		aux = arreglo_b[i*N+j];
		arreglo_b[i*N+j] = arreglo_b[j*N+i];
		arreglo_b[j*N+i] = aux;
	}

}

// copia B en C
__global__ void copiador(int* arreglo_b, int* arreglo_c, int N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N)
		arreglo_c[tid] = arreglo_b[tid];
}

// C += A
__global__ void sumador(int* arreglo_a, int* arreglo_c, int N){
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid < N)
		arreglo_c[tid] += arreglo_a[tid];
}

// C += A * B^t
__global__ void multiplicador(int* arreglo_a, int* arreglo_b_trans, int* arreglo_c, int N, int total_elem){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	int i = (int)tid / N; // columna
	int j = (int)tid % N; // fila

	int k;
	int cuenta = 0;
	if(tid < total_elem)
	{
		for (k=0; k< N; k++){
			cuenta += arreglo_a[i*N+k] * arreglo_b_trans[k*N+j];
		}
		arreglo_c[tid] += cuenta;
	}

}


void solucion_CPU(){
	int* arreglo_at;
	int* arreglo_bt;
	int* arreglo_a_por_b;
	int* arreglo_res;


	int numBytes = sizeof(int) * cant_elem; //bytes a alocar

	arreglo_at = (int *) malloc(numBytes);
	arreglo_bt = (int *) malloc(numBytes);
	arreglo_a_por_b = (int *) malloc(numBytes); // resultado de A * B^t
	arreglo_res = (int *) malloc(numBytes);

	double timetick;
	timetick = dwalltime();

	// guardamos en arreglo_bt y arreglo_at los datos que van a ser transpuestos
	for (int i = 0; i < cant_elem; ++i)
	{
		arreglo_bt[i] = arreglo_B[i];
		arreglo_at[i] = arreglo_A[i];
	}

	// calculamos la transpuesta de B
	for (int i = 0; i < (cant * (cant+1))/2; ++i)
	{
		int col = int((1+sqrtf(1+8*i))/2); // columna
		int row = i - (col*(col-1)/2); // fila

		int aux;
		if((col<cant) && (row<cant)){
			aux = arreglo_bt[col*cant+row];
			arreglo_bt[col*cant+row] = arreglo_bt[row*cant+col];
			arreglo_bt[row*cant+col] = aux;
		}
	}

	// calculamos la transpuesta de A
	for (int i = 0; i < (cant * (cant+1))/2; ++i)
	{
		int col = int((1+sqrtf(1+8*i))/2); // columna
		int row = i - (col*(col-1)/2); // fila

		int aux;
		if((col<cant) && (row<cant)){
			aux = arreglo_at[col*cant+row];
			arreglo_at[col*cant+row] = arreglo_at[row*cant+col];
			arreglo_at[row*cant+col] = aux;
		}
	}

	for (int i = 0; i < cant_elem; i++)
	{
		int col = i / cant; // columna
		int row = i % cant; // fila
		int mul = 0;
		for (int k=0; k< cant; k++){
			mul += arreglo_A[col*cant+k] * arreglo_bt[k*cant+row];
		}
		arreglo_a_por_b[i] = mul;
	}

	for (int i = 0; i < cant_elem; i++){
		arreglo_res[i] = 0;
	}
	// C = B + A * B^t + A^t
	for (int i = 0; i < cant_elem; i++){
		arreglo_res[i] += arreglo_B[i] + arreglo_a_por_b[i] + arreglo_at[i];
	}

	
	printf("-> Tiempo transcurrido en la CPU %f\n", dwalltime() - timetick);

	// printf("%s\n", "");
	// printf("%s\n", "Resultados CPU:");
	// for (int i = 0; i < cant_elem; i++){
	// 	printf("%d\n", arreglo_res[i]);
	// }




	free(arreglo_at);
	free(arreglo_bt);
	free(arreglo_a_por_b);
	free(arreglo_res);
}


int main(int argc, char** argv){
	int numBytes = sizeof(int) * cant_elem; //bytes a alocar

	arreglo_A = (int *) malloc(numBytes);
	arreglo_B = (int *) malloc(numBytes);
	arreglo_C = (int *) malloc(numBytes);

	// llenamos los arreglos
	init_CPU_array(arreglo_A, cant_elem);
	init_CPU_array(arreglo_B, cant_elem);
	init_CPU_array(arreglo_C, cant_elem);

	// allocamos memoria en la gpu
	hipMalloc(&d_arreglo_A, numBytes);
	hipMalloc(&d_arreglo_B, numBytes);
	hipMalloc(&d_arreglo_C, numBytes);

	// copiamos los datos de la cpu a la gpu
	hipMemcpy(d_arreglo_A, arreglo_A, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_arreglo_B, arreglo_B, numBytes, hipMemcpyHostToDevice);

	double timetick;
	timetick = dwalltime();

	dim3 miGrid1D(512,1);
	dim3 miBloque1D(512,1);
	dim3 miBloque1D_transposeador((cant * (cant + 1))/2,1); // (N*(N+1))/2
	

	// C = B
	copiador<<<miGrid1D, miBloque1D>>>(d_arreglo_B, d_arreglo_C, cant_elem);

	// B^t
	transposeador<<<miGrid1D, miBloque1D_transposeador>>>(d_arreglo_B, cant);

	// C += A * B^t
 	multiplicador <<<miGrid1D, miBloque1D>>>(d_arreglo_A, d_arreglo_B, d_arreglo_C, cant, cant_elem);

	// A^t
	transposeador<<<miGrid1D, miBloque1D_transposeador>>>(d_arreglo_A, cant);

	// C += A^t
	sumador<<<miGrid1D, miBloque1D>>>(d_arreglo_A, d_arreglo_C, cant_elem);

	// esperamos a que termine la ejecucion
 	hipDeviceSynchronize();
	printf("-> Tiempo transcurrido en la GPU %f\n", dwalltime() - timetick);

	// nos traemos los resultados de la gpu a la cpu
	hipMemcpy(arreglo_C, d_arreglo_C, numBytes, hipMemcpyDeviceToHost);

	// imprimimos los resultados
	// printf("%s\n", "");
	// printf("%s\n", "Resultados GPU:");
	// print_CPU_array(arreglo_C, cant_elem);

	solucion_CPU();

	// liberamos memoria
	free(arreglo_A);
	free(arreglo_B);
	free(arreglo_C);
	hipFree (d_arreglo_A);
	hipFree (d_arreglo_B);
	hipFree (d_arreglo_C);

}
