#include "hip/hip_runtime.h"
#include "stdio.h"




void printi(int i){
	printf("%d\n", i);
}


void init_CPU_array(int* array, int n){
	for(int i = 0; i < n; i++) {
		array[i] = 1;
	}
}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}
void print_CPU_matrix(int array[], int n){
    for(int i = 0; i < n; i++) {
        if(i % 16 == 0)
            printf("%s\n", "");

        printf("%d ", array[i]);
    }
}


// realiza la suma de determinantes
__global__ void sumador(int* arreglo, int* result, float N)
{
	__shared__ int compartida[10];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	compartida[threadIdx.x] = arreglo[tid];
	__syncthreads();
	for(int i=1; pow((float)2,(float)i-1) < N; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1);
		if(threadIdx.x < (N/acceso) && (threadIdx.x * acceso + offset) < (N - blockIdx.x * blockDim.x))
		{
				compartida[threadIdx.x * acceso] = compartida[threadIdx.x * acceso] + compartida[threadIdx.x * acceso + offset];
				compartida[threadIdx.x * acceso + offset] = 0;
				printf("%s\n", "TRABAJO");
				result[blockIdx.x] = compartida[0];

		}

		printf("%s\n", "");
		
	}

}





int* arreglo_suma1;
int* d_arreglo_suma1;

int* arreglo_result;
int* d_arreglo_suma2;

int main(int argc, char** argv){
	int N = 8;

	//##################################################################################
	//############################## INICIALIZACION ####################################

	arreglo_suma1 = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_suma1, N * sizeof(int));

	arreglo_result = (int*) malloc(N * sizeof(int));
	hipMalloc(&d_arreglo_suma2, N * sizeof(int));


	init_CPU_array(arreglo_suma1, N);
	hipMemcpy(d_arreglo_suma1, arreglo_suma1, N * sizeof(int), hipMemcpyHostToDevice);

	int threads_per_block = 10;
	int block_count = ceil((float)N / threads_per_block);

	//##################################################################################
	//################################ EJECUCIONES #####################################

	dim3 miGrid1D_1(block_count,1);
	dim3 miBloque1D_1(threads_per_block,1);
	sumador<<<miGrid1D_1, miBloque1D_1>>>(d_arreglo_suma1, d_arreglo_suma2, N);


	//###################################################################################
	//################################### READ BACK #####################################

	hipMemcpy(arreglo_result, d_arreglo_suma2, N * sizeof(int), hipMemcpyDeviceToHost);

	printf("%s\n", "RESULTADO DE LA SUMA:");
	print_CPU_matrix(arreglo_result, N);

	free(arreglo_suma1);
	hipFree (d_arreglo_suma1);

	free(arreglo_result);
	hipFree (d_arreglo_suma2);

}