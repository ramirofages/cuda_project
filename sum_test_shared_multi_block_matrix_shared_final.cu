#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
#define threads_per_block 32




void printi(int i){
	printf("%d\n", i);
}


void init_CPU_array(int* arreglo_b, int n){
	for(int i=0; i< n; i++)
	{
		int valor = 1;
		arreglo_b[(i*16) + 0] = valor;
		arreglo_b[(i*16) + 1] = valor;
		arreglo_b[(i*16) + 2] = valor;
		arreglo_b[(i*16) + 3] = valor;
		arreglo_b[(i*16) + 4] = valor;
		arreglo_b[(i*16) + 5] = valor;
		arreglo_b[(i*16) + 6] = valor;
		arreglo_b[(i*16) + 7] = valor;
		arreglo_b[(i*16) + 8] = valor;
		arreglo_b[(i*16) + 9] = valor;
		arreglo_b[(i*16) + 10] = valor;
		arreglo_b[(i*16) + 11] = valor;
		arreglo_b[(i*16) + 12] = valor;
		arreglo_b[(i*16) + 13] = valor;
		arreglo_b[(i*16) + 14] = valor;
		arreglo_b[(i*16) + 15] = valor;

	}


}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}

void print_CPU_matrix(int array[], int n){
    for(int i = 0; i < n; i++) {
        if(i % 16 == 0)
            printf("%s\n", "");

        printf("%d ", array[i]);
    }
}


__global__ void sumador(int* arreglo, int* result, float N)
{
	__shared__ int compartida[threads_per_block * 16];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid > N * 16)
		return;

	compartida[threadIdx.x] = arreglo[tid];
	__syncthreads();
	for(int i=1; pow((float)2,(float)i-1) < N; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1);

		int t_id = (threadIdx.x/16) * 16;
		int new_access = t_id * acceso + threadIdx.x % 16 ;
		int new_offset = new_access + offset * 16;

		if(t_id < ((float)threads_per_block*16/acceso) && (new_offset  < (threads_per_block*16)))
		{
				

				compartida[new_access] = compartida[new_access] + compartida[new_offset];
				compartida[new_offset] = 0;
				// printf("GRUPO: %d - ITERACION: %d - TID %d - ACCESO: %d - OFFSET %d - REMAINING: %d \n", blockIdx.x,
				// 														i, 			tid, 	new_access , new_offset, threadIdx.x * acceso + offset);
		}
		__syncthreads();


	}

	//el primer thread de cada grupo guarda el resultado
	if(threadIdx.x < 16)
		result[blockIdx.x * 16 + threadIdx.x] = compartida[threadIdx.x];

}





int* arreglo_suma1;
int* d_arreglo_suma1;

int* arreglo_result;
int* d_arreglo_suma2;

int main(int argc, char** argv){

	int N = 1100000;
	//##################################################################################
	//############################## INICIALIZACION ####################################
	int byte_size = N * sizeof(int) * 16;
	arreglo_suma1 = (int*) malloc(byte_size);
	hipMalloc(&d_arreglo_suma1, byte_size);

	arreglo_result = (int*) malloc(byte_size);
	hipMalloc(&d_arreglo_suma2, byte_size);


	init_CPU_array(arreglo_suma1, N);
	hipMemcpy(d_arreglo_suma1, arreglo_suma1, byte_size, hipMemcpyHostToDevice);


	//##################################################################################
	//################################ EJECUCIONES #####################################


	dim3 miBloque1D_1(threads_per_block *16,1);
	for(int i=1; pow(threads_per_block, i-1) < N; i++)
	{
		int remaining_elements = ceil((float)N/pow(threads_per_block, i-1));
		int block_count = ceil((float)N/pow(threads_per_block * 16, i-1));

		dim3 miGrid1D_1(remaining_elements,1);
		sumador<<<miGrid1D_1, miBloque1D_1>>>(d_arreglo_suma1, d_arreglo_suma2, remaining_elements);
		hipDeviceSynchronize();
		printf("ERROR %s\n", hipGetErrorString(hipGetLastError()));
		int* tmp = d_arreglo_suma1;
		d_arreglo_suma1 = d_arreglo_suma2;
		d_arreglo_suma2 = tmp;

		printf("elementos restantes: %d \n ", remaining_elements);
		printf("block_count: %d \n ", block_count);
		printf("\n ", "");


	}


	//##################################################################################
	//################################### READ BACK #####################################

	hipMemcpy(arreglo_result, d_arreglo_suma1, N * sizeof(int) * 16, hipMemcpyDeviceToHost);

	printf("%s\n", "RESULTADO DE LA SUMA:");
	print_CPU_matrix(arreglo_result, 1 * 16);

	free(arreglo_suma1);
	hipFree (d_arreglo_suma1);

	free(arreglo_result);
	hipFree (d_arreglo_suma2);

}