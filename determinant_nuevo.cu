#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"


//#include <sys/time.h>
//#include <sys/resource.h>


// double dwalltime(){
//         double sec;
//         struct timeval tv;

//         gettimeofday(&tv,NULL);
//         sec = tv.tv_sec + tv.tv_usec/1000000.0;
//         return sec;
// }


int N = 3;


// arreglos usados como matrices

int* arreglo_A;
int* arreglo_B;
int* arreglo_C;

int* suma_det;

int* d_arreglo_A;
int* d_arreglo_B;
int* d_arreglo_C;

int* arreglo_determinantes;
int* d_arreglo_determinantes;


void printi(int i){
	printf("%d\n", i);
}

void init_CPU_matrices_array(int* arreglo_b, int n){
	for(int i=0; i< N; i++)
	{
		arreglo_b[(i*16) + 0] = 1;
		arreglo_b[(i*16) + 1] = 2;
		arreglo_b[(i*16) + 2] = 0;
		arreglo_b[(i*16) + 3] = 0;
		arreglo_b[(i*16) + 4] = 0;
		arreglo_b[(i*16) + 5] = 1;
		arreglo_b[(i*16) + 6] = 0;
		arreglo_b[(i*16) + 7] = 0;
		arreglo_b[(i*16) + 8] = 0;
		arreglo_b[(i*16) + 9] = 0;
		arreglo_b[(i*16) + 10] = 1;
		arreglo_b[(i*16) + 11] = 0;
		arreglo_b[(i*16) + 12] = 0;
		arreglo_b[(i*16) + 13] = 0;
		arreglo_b[(i*16) + 14] = 0;
		arreglo_b[(i*16) + 15] = 1;
	}

}
void init_CPU_array(int* array, int n){
	for(int i = 0; i < n; i++) {
		array[i] = i;
	}
}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}

void print_CPU_array_double(double array[], int n){
	for(int i = 0; i < n; i++) {
		printf("%f\n", array[i]);
	}
}




__global__ void determinanteador(int* arreglo_b, int* arreglo_a, int N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	

	int mat[9];
	mat[0] = arreglo_b[(tid * 16) + 5];
	mat[1] = arreglo_b[(tid * 16) + 6];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 9];
	mat[4] = arreglo_b[(tid * 16) + 10];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 13];
	mat[7] = arreglo_b[(tid * 16) + 14];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	float det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	float det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	float det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result0 = det0 - det1 + det2;
	result0 *= arreglo_b[0];	

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 6];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 10];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 14];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result1 = det0 - det1 + det2;
	result1 *= arreglo_b[1];

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 5];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 9];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 13];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result2 = det0 - det1 + det2;
	result2 *= arreglo_b[2];

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 5];
	mat[2] = arreglo_b[(tid * 16) + 6];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 9];
	mat[5] = arreglo_b[(tid * 16) + 10];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 13];
	mat[8] = arreglo_b[(tid * 16) + 14];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result3 = det0 - det1 + det2;
	result3 *= arreglo_b[3];

	float result_total = result0 - result1 + result2 - result3;
	arreglo_a[tid] = result_total;
	
}

// realiza la suma total de forma paralela, aumentando el offset en cada ejecucion
__global__ void sumador(int* arreglo, int offset, int N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid < N){

		if( (tid & ( (offset * 2) -1)) == 0 && ( (tid+offset) < N))
		{

			arreglo[tid] = arreglo[tid] + arreglo[tid + offset];

		}


	}

}


// realiza la suma de n matrices
__global__ void sumador_2(int* arreglo, int offset, int N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	shared int[10] compartido;
	compartido[threadIdx.x] = arreglo1[tid];
	groupSynchronize();

	for(int i=1; i< 32/16; i*=2)
	{
		if(tid < N*16){

			if( (( (tid / 16) & (offset*2)-1 ) == 0) && ( (tid+offset*16) < N*16))
			{
				arreglo[tid] = arreglo[tid] + arreglo[tid + offset * 16];
			}
		}
	}
}


// suma las determinantes de forma paralela y sin divergencia
__global__ void sumador_3(int* arreglo, int acceso, int offset, int i, float N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	// if(tid < N/acceso)
	// {
	// 	printf("%d\n", arreglo[tid * acceso]);
	// 	printf("%d\n", arreglo[tid * acceso + offset]);
	// }
	
	if(tid < N/acceso)
	{
			arreglo[tid * acceso] = arreglo[tid * acceso] + arreglo[tid * acceso + offset];
			arreglo[tid * acceso + offset] = 0; //solo para debugear
			printf("%s\n", "TRABAJO");

	}
}

// sumador sin optimizar :C
__global__ void sumador_4(int* arreglo1, int* arreglo2, int offset, int N, int threads_per_block){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	shared int[10] compartido;
	compartido[threadIdx.x] = arreglo1[tid];
	groupSynchronize();
	if(blockIdx.x < N/threads_per_block)
	{
		for(int i=1; i< 10; i*=2)
		{
		

				if( (threadIdx.x & ( (i * 2) -1)) == 0 && ( (threadIdx.x+i) < N))
				{

					compartido[threadIdx.x] = compartido[threadIdx.x] + compartido[threadIdx.x + ia];

				}


			}
	}

	arreglo2[blockIdx.x] = compartido[0];


}

void multiplicar(double num, int* mat, double mat_res[16])
{
	for(int i=0; i< 16; i++)
	{
		mat_res[i] = mat[i] * num;
	}
}





int main(int argc, char** argv){
	int numBytes = sizeof(int) * N; //bytes a alocar
	int numBytesDeterminantes = sizeof(int) * N; //bytes a alocar
	int num = sizeof(int)*16;

	suma_det = (int *) malloc(sizeof(int)); 
	arreglo_A = (int *) malloc(numBytes);
	arreglo_B = (int *) malloc(numBytes * 16);
	arreglo_C = (int *) malloc(num);


	arreglo_determinantes = (int*) malloc(numBytesDeterminantes);
	hipMalloc(&d_arreglo_determinantes, numBytesDeterminantes);

	init_CPU_array(arreglo_determinantes, N);
	hipMemcpy(d_arreglo_determinantes, arreglo_determinantes, numBytesDeterminantes, hipMemcpyHostToDevice);

	dim3 miGrid1D_suma_determinantes(1,1);
	dim3 miBloque1D_suma_determinantes(N,1);


	for(int i=1; pow(2,i-1) < N; i++)
	{
		sumador_3<<<miGrid1D_suma_determinantes, 
								miBloque1D_suma_determinantes>>>(d_arreglo_determinantes, pow(2,i), pow(2, i-1), i, N);
		hipDeviceSynchronize();
		
		printf("%s\n", " ");

	}

	hipMemcpy(arreglo_determinantes, d_arreglo_determinantes, numBytesDeterminantes, hipMemcpyDeviceToHost);

	printf("%s\n", "SUMA DETERMINANTES RESULT:");
	print_CPU_array(arreglo_determinantes, N);


	float promedio_determinantes = (float)arreglo_determinantes[0] / N;

	printf("%s\n", "PROMEDIO DETERMINANTES:");
	printf("%f\n", promedio_determinantes);










	//double timetick;

	// llenamos los arreglos

	init_CPU_matrices_array(arreglo_B, N);


	// allocamos memoria en la gpu

	hipMalloc(&d_arreglo_B, numBytes * 16);
	hipMalloc(&d_arreglo_C, numBytes);
	hipMalloc(&d_arreglo_A, numBytes);



	// copiamos los datos de la cpu a la gpu

	hipMemcpy(d_arreglo_B, arreglo_B, numBytes * 16, hipMemcpyHostToDevice);

	dim3 miGrid1D(1,1);
	dim3 miBloque1D(16 * N,1);
	dim3 miBloque1D_determinanteador(N,1);

	// timetick = dwalltime();


	// si tenemos 10 matrises, 10 determinants
	determinanteador<<<miGrid1D, miBloque1D_determinanteador>>>(d_arreglo_B, d_arreglo_A, N);

	for(int i=1; i < N; i*= 2){
		sumador<<<miGrid1D, miBloque1D>>>(d_arreglo_A, i, N);
		hipDeviceSynchronize();


	}



	hipMemcpy(suma_det, d_arreglo_A, sizeof(int), hipMemcpyDeviceToHost);

	double promedio_det = (*suma_det) / N;

	// Sumamos todos los elementos para el promedio, el resultado queda almacenado en la primer posicion
	for(int i=1; i < N; i*= 2){
		sumador_2<<<miGrid1D, miBloque1D>>>(d_arreglo_B, i, N);
		hipDeviceSynchronize();


	}
	
	// nos traemos los resultados de la gpu a la cpu
	hipMemcpy(arreglo_C, d_arreglo_B, num, hipMemcpyDeviceToHost);

	// printf("-> Tiempo transcurrido en la GPU %f\n", dwalltime() - timetick);


	double mat_res[16];
	multiplicar(promedio_det, arreglo_C, mat_res);

	//imprimimos los resultados
	// printf("%s\n", "");
	// printf("%s\n", "Promedio determinante:");

	// printf("%lf\n", promedio_det);

	

	// printf("%s\n", "MATRIZ RESULTANTE: ");
	// print_CPU_array_double(mat_res, 16);

	
	// Sumamos todos los elementos para el promedio, el resultado queda almacenado en la primer posicion









	int txb=10;

	dim3 miGrid1D_suma_4(2,1);
	dim3 miBloque1D_suma_4(txb,1);
	for(int i=1; i <= cant_elem/txb; i*= 2){
		sumador_4<<<miGrid1D_suma_4, miBloque1D_suma_4>>>(d_arreglo_suma1, d_arreglo_suma2, i, cant_elem, txb);
		hipDeviceSynchronize();

		int* temp = d_arreglo_suma1;
		d_arreglo_suma1 = d_arreglo_suma2;
		d_arreglo_suma2 = tmp;
	}


	free(arreglo_determinantes);
	hipFree (d_arreglo_determinantes);
	
	free(arreglo_A);
	free(arreglo_B);
	free(arreglo_C);
	free(suma_det);
	

	hipFree (d_arreglo_A);
	hipFree (d_arreglo_B);
	hipFree (d_arreglo_C);

}
