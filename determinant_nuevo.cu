#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"


//#include <sys/time.h>
//#include <sys/resource.h>


// double dwalltime(){
//         double sec;
//         struct timeval tv;

//         gettimeofday(&tv,NULL);
//         sec = tv.tv_sec + tv.tv_usec/1000000.0;
//         return sec;
// }




int* arreglo_result;
int* mat_result;
int* suma_det;



int* arreglo_A;
int* d_arreglo_A;

int* arreglo_B;
int* d_arreglo_B;

int* mat_A;
int* d_mat_A;

int* mat_B;
int* d_mat_B;

int* arreglo_determinantes;
int* d_arreglo_determinantes;


void printi(int i){
	printf("%d\n", i);
}

void init_CPU_matrices_array(int* arreglo, int n){
	for(int i=0; i< n; i++)
	{
		//int valor = 1;
		arreglo[(i*16) + 0] = 5;
		arreglo[(i*16) + 1] = 5;
		arreglo[(i*16) + 2] = 3;
		arreglo[(i*16) + 3] = 4;
		arreglo[(i*16) + 4] = 12;
		arreglo[(i*16) + 5] = 3;
		arreglo[(i*16) + 6] = 4;
		arreglo[(i*16) + 7] = 5;
		arreglo[(i*16) + 8] = 6;
		arreglo[(i*16) + 9] = 7;
		arreglo[(i*16) + 10] = 8;
		arreglo[(i*16) + 11] = 9;
		arreglo[(i*16) + 12] = 10;
		arreglo[(i*16) + 13] = 1;
		arreglo[(i*16) + 14] = 2;
		arreglo[(i*16) + 15] = 3;

	}

}
void init_CPU_array(int* array, int n){
	for(int i = 0; i < n; i++) {
		array[i] = i;
	}
}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}
void print_CPU_matrix(int array[], int n){
    for(int i = 0; i < n; i++) {
        if(i % 16 == 0)
            printf("%s\n", "");

        printf("%d ", array[i]);
    }
}



__global__ void determinanteador(int* arreglo_b, int* arreglo_a, int N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(tid > N)
		return;


	int mat[9];
	mat[0] = arreglo_b[(tid * 16) + 5];
	mat[1] = arreglo_b[(tid * 16) + 6];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 9];
	mat[4] = arreglo_b[(tid * 16) + 10];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 13];
	mat[7] = arreglo_b[(tid * 16) + 14];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	float det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	float det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	float det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result0 = det0 - det1 + det2;
	result0 *= arreglo_b[0];	

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 6];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 10];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 14];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result1 = det0 - det1 + det2;
	result1 *= arreglo_b[1];

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 5];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 9];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 13];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result2 = det0 - det1 + det2;
	result2 *= arreglo_b[2];

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 5];
	mat[2] = arreglo_b[(tid * 16) + 6];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 9];
	mat[5] = arreglo_b[(tid * 16) + 10];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 13];
	mat[8] = arreglo_b[(tid * 16) + 14];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result3 = det0 - det1 + det2;
	result3 *= arreglo_b[3];

	float result_total = result0 - result1 + result2 - result3;
	arreglo_a[tid] = result_total;
	
}




int main(int argc, char** argv){

	int N = 520;
	int threads_per_block_determinanteador = 512;


	int numBytesMatrices = sizeof(int) * N * 16; //bytes a alocar
	int numBytesDeterminantes = sizeof(int) * N; //bytes a alocar

	//##################################################################################
	//############################## INICIALIZACION ####################################

	suma_det = (int *) malloc(sizeof(int)); 

	arreglo_A = (int *) malloc(numBytesDeterminantes);	
	arreglo_B = (int *) malloc(numBytesDeterminantes);	


	mat_A = (int *) malloc(numBytesMatrices);
	mat_B = (int *) malloc(numBytesMatrices);

	arreglo_determinantes = (int*) malloc(numBytesDeterminantes);

	hipMalloc(&d_arreglo_determinantes, numBytesDeterminantes);
	hipMalloc(&d_arreglo_A, numBytesDeterminantes);
	hipMalloc(&d_arreglo_B, numBytesDeterminantes);

	hipMalloc(&d_mat_A, numBytesMatrices);
	hipMalloc(&d_mat_B, numBytesMatrices);

	init_CPU_matrices_array(mat_A, N);
	hipMemcpy(d_mat_A, mat_A, numBytesMatrices, hipMemcpyHostToDevice);


	//##################################################################################
	//################################ EJECUCIONES #####################################


	//################################ DETERMINANTE ####################################
	dim3 miGrid1D_suma_determinantes(ceil((float)N/threads_per_block_determinanteador),1);
	dim3 miBloque1D_suma_determinantes(threads_per_block_determinanteador,1);

	determinanteador<<<miGrid1D_suma_determinantes,miBloque1D_suma_determinantes>>>(d_mat_A, d_arreglo_determinantes, N);
	hipDeviceSynchronize();
	// printf("ERROR %s\n", hipGetErrorString(hipGetLastError()));
	hipMemcpy(arreglo_determinantes, d_arreglo_determinantes, numBytesDeterminantes, hipMemcpyDeviceToHost);
	print_CPU_array(arreglo_determinantes, N);








	// //double timetick;

	// // llenamos los arreglos

	// init_CPU_matrices_array(arreglo_B, N);


	// // allocamos memoria en la gpu

	// hipMalloc(&d_arreglo_B, numBytes * 16);
	// hipMalloc(&d_arreglo_C, numBytes);
	// hipMalloc(&d_arreglo_A, numBytes);



	// // copiamos los datos de la cpu a la gpu

	// hipMemcpy(d_arreglo_B, arreglo_B, numBytes * 16, hipMemcpyHostToDevice);

	// dim3 miGrid1D(1,1);
	// dim3 miBloque1D(16 * N,1);
	// dim3 miBloque1D_determinanteador(N,1);

	// // timetick = dwalltime();


	// // si tenemos 10 matrises, 10 determinants
	// determinanteador<<<miGrid1D, miBloque1D_determinanteador>>>(d_arreglo_B, d_arreglo_A, N);

	// for(int i=1; i < N; i*= 2){
	// 	sumador<<<miGrid1D, miBloque1D>>>(d_arreglo_A, i, N);
	// 	hipDeviceSynchronize();


	// }



	// hipMemcpy(suma_det, d_arreglo_A, sizeof(int), hipMemcpyDeviceToHost);

	// double promedio_det = (*suma_det) / N;

	// // Sumamos todos los elementos para el promedio, el resultado queda almacenado en la primer posicion
	// for(int i=1; i < N; i*= 2){
	// 	sumador_2<<<miGrid1D, miBloque1D>>>(d_arreglo_B, i, N);
	// 	hipDeviceSynchronize();


	// }
	
	// // nos traemos los resultados de la gpu a la cpu
	// hipMemcpy(arreglo_C, d_arreglo_B, num, hipMemcpyDeviceToHost);

	// // printf("-> Tiempo transcurrido en la GPU %f\n", dwalltime() - timetick);


	// double mat_res[16];
	// multiplicar(promedio_det, arreglo_C, mat_res);

	// //imprimimos los resultados
	// // printf("%s\n", "");
	// // printf("%s\n", "Promedio determinante:");

	// // printf("%lf\n", promedio_det);

	

	// // printf("%s\n", "MATRIZ RESULTANTE: ");
	// // print_CPU_array_double(mat_res, 16);

	
	// // Sumamos todos los elementos para el promedio, el resultado queda almacenado en la primer posicion









	// int txb=10;

	// dim3 miGrid1D_suma_4(2,1);
	// dim3 miBloque1D_suma_4(txb,1);
	// for(int i=1; i <= cant_elem/txb; i*= 2){
	// 	sumador_4<<<miGrid1D_suma_4, miBloque1D_suma_4>>>(d_arreglo_suma1, d_arreglo_suma2, i, cant_elem, txb);
	// 	hipDeviceSynchronize();

	// 	int* temp = d_arreglo_suma1;
	// 	d_arreglo_suma1 = d_arreglo_suma2;
	// 	d_arreglo_suma2 = tmp;
	// }


	free(arreglo_determinantes);
	free(suma_det);

	free(arreglo_A);
	free(arreglo_B);

	free(mat_A);
	free(mat_B);
	

	hipFree (d_arreglo_A);
	hipFree (d_arreglo_B);
	hipFree (d_arreglo_determinantes);

	hipFree (d_mat_A);
	hipFree (d_mat_B);


}


