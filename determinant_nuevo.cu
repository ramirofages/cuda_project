#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"

#define threads_per_block 512
#define threads_per_block_matrix 32 // 32*16 = 512


//#include <sys/time.h>
//#include <sys/resource.h>


// double dwalltime(){
//         double sec;
//         struct timeval tv;

//         gettimeofday(&tv,NULL);
//         sec = tv.tv_sec + tv.tv_usec/1000000.0;
//         return sec;
// }




int* arreglo_result;
int* mat_result;
int* suma_det;



int* arreglo_A;
int* d_arreglo_A;

int* arreglo_B;
int* d_arreglo_B;

int* mat_A;
int* d_mat_A;

int* mat_B;
int* d_mat_B;

int* arreglo_determinantes;
int* d_arreglo_determinantes;


void printi(int i){
	printf("%d\n", i);
}

void init_CPU_matrices_array(int* arreglo, int n){
	for(int i=0; i< n; i++)
	{
		//int valor = 1;
		arreglo[(i*16) + 0] = 1;
		arreglo[(i*16) + 1] = 0;
		arreglo[(i*16) + 2] = 0;
		arreglo[(i*16) + 3] = 0;
		arreglo[(i*16) + 4] = 0;
		arreglo[(i*16) + 5] = 1;
		arreglo[(i*16) + 6] = 0;
		arreglo[(i*16) + 7] = 0;
		arreglo[(i*16) + 8] = 0;
		arreglo[(i*16) + 9] = 0;
		arreglo[(i*16) + 10] = 1;
		arreglo[(i*16) + 11] = 0;
		arreglo[(i*16) + 12] = 0;
		arreglo[(i*16) + 13] = 0;
		arreglo[(i*16) + 14] = 0;
		arreglo[(i*16) + 15] = 1;

	}

}
void init_CPU_array(int* array, int n){
	for(int i = 0; i < n; i++) {
		array[i] = i;
	}
}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}

void print_CPU_matrix(int array[], int n){
    for(int i = 0; i < n; i++) {
        if(i % 16 == 0)
            printf("%s\n", "");

        printf("%d ", array[i]);
    }
}



__global__ void determinanteador(int* arreglo_b, int* arreglo_a, int N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(tid > N)
		return;


	int mat[9];
	mat[0] = arreglo_b[(tid * 16) + 5];
	mat[1] = arreglo_b[(tid * 16) + 6];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 9];
	mat[4] = arreglo_b[(tid * 16) + 10];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 13];
	mat[7] = arreglo_b[(tid * 16) + 14];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	float det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	float det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	float det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result0 = det0 - det1 + det2;
	result0 *= arreglo_b[0];	

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 6];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 10];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 14];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result1 = det0 - det1 + det2;
	result1 *= arreglo_b[1];

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 5];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 9];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 13];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result2 = det0 - det1 + det2;
	result2 *= arreglo_b[2];

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 5];
	mat[2] = arreglo_b[(tid * 16) + 6];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 9];
	mat[5] = arreglo_b[(tid * 16) + 10];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 13];
	mat[8] = arreglo_b[(tid * 16) + 14];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result3 = det0 - det1 + det2;
	result3 *= arreglo_b[3];

	float result_total = result0 - result1 + result2 - result3;
	arreglo_a[tid] = result_total;
	
}

// realiza la suma de determinantes
__global__ void sumador_determinantes(int* arreglo, int* result, float N)
{
	__shared__ int compartida[threads_per_block];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid > N)
		return;
	
	compartida[threadIdx.x] = arreglo[tid];
	__syncthreads();
	for(int i=1; pow((float)2,(float)i-1) < threads_per_block; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1);
		if(threadIdx.x < ((float)threads_per_block/acceso) && (threadIdx.x * acceso + offset) < (N - blockIdx.x * blockDim.x))
		{
				compartida[threadIdx.x * acceso] = compartida[threadIdx.x * acceso] + compartida[threadIdx.x * acceso + offset];
				compartida[threadIdx.x * acceso + offset] = 0;
		}

	}

	//el primer thread de cada grupo guarda el resultado
	if(threadIdx.x == 0)
		result[blockIdx.x] = compartida[0];

}

__global__ void sumador_matrices(int* arreglo, int* result, float N)
{
	__shared__ int compartida[threads_per_block_matrix * 16];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid > N * 16)
		return;

	compartida[threadIdx.x] = arreglo[tid];
	__syncthreads();
	for(int i=1; pow((float)2,(float)i-1) < N; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1);

		int t_id = (threadIdx.x/16) * 16;
		int new_access = t_id * acceso + threadIdx.x % 16 ;
		int new_offset = new_access + offset * 16;

		if(t_id < ((float)threads_per_block_matrix*16/acceso) && (new_offset  < (threads_per_block_matrix*16)))
		{
				

				compartida[new_access] = compartida[new_access] + compartida[new_offset];
				compartida[new_offset] = 0;
				// printf("GRUPO: %d - ITERACION: %d - TID %d - ACCESO: %d - OFFSET %d - REMAINING: %d \n", blockIdx.x,
				// 														i, 			tid, 	new_access , new_offset, threadIdx.x * acceso + offset);
		}


	}

	//el primer thread de cada grupo guarda el resultado
	if(threadIdx.x < 16)
		result[blockIdx.x * 16 + threadIdx.x] = compartida[threadIdx.x];

}

int main(int argc, char** argv){

	int N = 256;

	int numBytesMatrices = sizeof(int) * N * 16; //bytes a alocar
	int numBytesDeterminantes = sizeof(int) * N; //bytes a alocar

	//##################################################################################
	//############################## INICIALIZACION ####################################

	suma_det = (int *) malloc(sizeof(int)); 

	arreglo_A = (int *) malloc(numBytesDeterminantes);	
	arreglo_B = (int *) malloc(numBytesDeterminantes);	


	mat_A = (int *) malloc(numBytesMatrices);
	mat_B = (int *) malloc(numBytesMatrices);

	arreglo_determinantes = (int*) malloc(numBytesDeterminantes);

	hipMalloc(&d_arreglo_determinantes, numBytesDeterminantes);
	hipMalloc(&d_arreglo_A, numBytesDeterminantes);
	hipMalloc(&d_arreglo_B, numBytesDeterminantes);

	hipMalloc(&d_mat_A, numBytesMatrices);
	hipMalloc(&d_mat_B, numBytesMatrices);

	init_CPU_matrices_array(mat_A, N);
	hipMemcpy(d_mat_A, mat_A, numBytesMatrices, hipMemcpyHostToDevice);


	//##################################################################################
	//################################ EJECUCIONES #####################################


	//################################ DETERMINANTE ####################################
	dim3 miGrid1D_determinanteador(ceil((float)N/threads_per_block),1);
	dim3 miBloque1D_determinanteador(threads_per_block,1);

	determinanteador<<<miGrid1D_determinanteador,miBloque1D_determinanteador>>>(d_mat_A, d_arreglo_A, N);
	hipDeviceSynchronize();
	// printf("ERROR %s\n", hipGetErrorString(hipGetLastError()));
	// hipMemcpy(arreglo_determinantes, d_arreglo_A, numBytesDeterminantes, hipMemcpyDeviceToHost);
	// print_CPU_matrix(arreglo_determinantes, N);



	//############################# SUMADOR DETERMINANTE ###############################

	dim3 miBloque1D_sumador(threads_per_block,1);
	for(int i=1; pow(threads_per_block, i-1) < N; i++)
	{
		int remaining_elements = ceil((float)N/pow(threads_per_block, i-1));
		dim3 miGrid1D_sumador(remaining_elements,1);
		sumador_determinantes<<<miGrid1D_sumador, miBloque1D_sumador>>>(d_arreglo_A, d_arreglo_B, remaining_elements);
		hipDeviceSynchronize();
		// printf("ERROR: %s\n", hipGetErrorString(hipGetLastError()));

		int* tmp = d_arreglo_A;
		d_arreglo_A = d_arreglo_B;
		d_arreglo_B = tmp;
	}

	// hipMemcpy(arreglo_determinantes, d_arreglo_A, sizeof(int) * N, hipMemcpyDeviceToHost);
	// print_CPU_matrix(arreglo_determinantes, N);

	//############################## SUMADOR MATRICES ##################################

	dim3 miBloque1D_sumador_mat(threads_per_block_matrix *16,1);
	for(int i=1; pow(threads_per_block_matrix, i-1) < N; i++)
	{
		int remaining_elements = ceil((float)N/pow(threads_per_block_matrix, i-1));
		int block_count = ceil((float)N/pow(threads_per_block_matrix * 16, i-1));

		dim3 miGrid1D_sumador_mat(remaining_elements,1);
		sumador_matrices<<<miGrid1D_sumador_mat, miBloque1D_sumador_mat>>>(d_mat_A, d_mat_B, remaining_elements);
		hipDeviceSynchronize();
		// printf("ERROR %s\n", hipGetErrorString(hipGetLastError()));
		int* tmp = d_mat_A;
		d_mat_A = d_mat_B;
		d_mat_B = tmp;

	}

	//############################### READ BACK ########################################

	// PROMEDIO
	hipMemcpy(suma_det, d_arreglo_A, sizeof(int), hipMemcpyDeviceToHost);
	double promedio_det = (float)(*suma_det) / N;
	printf("PROMEDIO: %lf\n", promedio_det);
	
	// SUMA DE MATRICES
	hipMemcpy(mat_B, d_mat_A, 16 * sizeof(int), hipMemcpyDeviceToHost);


	for(int i=0; i< 16; i++)
		mat_B[i] *= (int)promedio_det;


	printf("%s\n", "RESULTADO:");
	print_CPU_matrix(mat_B, 16);



	free(arreglo_determinantes);
	free(suma_det);

	free(arreglo_A);
	free(arreglo_B);

	free(mat_A);
	free(mat_B);
	

	hipFree (d_arreglo_A);
	hipFree (d_arreglo_B);
	hipFree (d_arreglo_determinantes);

	hipFree (d_mat_A);
	hipFree (d_mat_B);


}


