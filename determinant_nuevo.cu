#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "stdio.h"
#include <time.h>
#define threads_per_block 512
#define threads_per_block_matrix 32 // 32*16 = 512


//#include <sys/time.h>
//#include <sys/resource.h>


// double dwalltime(){
//         double sec;
//         struct timeval tv;

//         gettimeofday(&tv,NULL);
//         sec = tv.tv_sec + tv.tv_usec/1000000.0;
//         return sec;
// }




int* arreglo_result;
int* mat_result;
int* suma_det;



int* arreglo_A;
int* d_arreglo_A;

int* arreglo_B;
int* d_arreglo_B;

int* mat_A;
int* d_mat_A;

int* mat_B;
int* d_mat_B;

int* arreglo_determinantes;
int* d_arreglo_determinantes;


void printi(int i){
	printf("%d\n", i);
}

void init_CPU_matrices_array(int* arreglo, int n){
	for(int i=0; i< n; i++)
	{
		//int valor = 1;
		arreglo[(i*16) + 0] = 1;
		arreglo[(i*16) + 1] = 0;
		arreglo[(i*16) + 2] = 0;
		arreglo[(i*16) + 3] = 0;
		arreglo[(i*16) + 4] = 0;
		arreglo[(i*16) + 5] = 1;
		arreglo[(i*16) + 6] = 0;
		arreglo[(i*16) + 7] = 0;
		arreglo[(i*16) + 8] = 0;
		arreglo[(i*16) + 9] = 0;
		arreglo[(i*16) + 10] = 1;
		arreglo[(i*16) + 11] = 0;
		arreglo[(i*16) + 12] = 0;
		arreglo[(i*16) + 13] = 0;
		arreglo[(i*16) + 14] = 0;
		arreglo[(i*16) + 15] = 1;

	}

}
void init_CPU_array(int* array, int n){
	for(int i = 0; i < n; i++) {
		array[i] = i;
	}
}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}

void print_CPU_matrix(int array[], int n){
    for(int i = 0; i < n; i++) {
        if(i % 16 == 0)
            printf("%s\n", "");

        printf("%d ", array[i]);
    }
}



__global__ void determinanteador(int* arreglo_b, int* arreglo_a, int N){

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(tid > N)
		return;


	int mat[9];
	mat[0] = arreglo_b[(tid * 16) + 5];
	mat[1] = arreglo_b[(tid * 16) + 6];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 9];
	mat[4] = arreglo_b[(tid * 16) + 10];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 13];
	mat[7] = arreglo_b[(tid * 16) + 14];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	float det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	float det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	float det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result0 = det0 - det1 + det2;
	result0 *= arreglo_b[0];	

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 6];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 10];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 14];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result1 = det0 - det1 + det2;
	result1 *= arreglo_b[1];

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 5];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 9];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 13];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result2 = det0 - det1 + det2;
	result2 *= arreglo_b[2];

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 5];
	mat[2] = arreglo_b[(tid * 16) + 6];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 9];
	mat[5] = arreglo_b[(tid * 16) + 10];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 13];
	mat[8] = arreglo_b[(tid * 16) + 14];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result3 = det0 - det1 + det2;
	result3 *= arreglo_b[3];

	float result_total = result0 - result1 + result2 - result3;
	arreglo_a[tid] = result_total;
	
}

void determinanteador_CPU(int* arreglo_b, int* arreglo_a, int tid){


	int mat[9];
	mat[0] = arreglo_b[(tid * 16) + 5];
	mat[1] = arreglo_b[(tid * 16) + 6];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 9];
	mat[4] = arreglo_b[(tid * 16) + 10];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 13];
	mat[7] = arreglo_b[(tid * 16) + 14];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	float det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	float det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	float det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result0 = det0 - det1 + det2;
	result0 *= arreglo_b[0];	

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 6];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 10];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 14];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result1 = det0 - det1 + det2;
	result1 *= arreglo_b[1];

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 5];
	mat[2] = arreglo_b[(tid * 16) + 7];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 9];
	mat[5] = arreglo_b[(tid * 16) + 11];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 13];
	mat[8] = arreglo_b[(tid * 16) + 15];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result2 = det0 - det1 + det2;
	result2 *= arreglo_b[2];

	mat[0] = arreglo_b[(tid * 16) + 4];
	mat[1] = arreglo_b[(tid * 16) + 5];
	mat[2] = arreglo_b[(tid * 16) + 6];
	mat[3] = arreglo_b[(tid * 16) + 8];
	mat[4] = arreglo_b[(tid * 16) + 9];
	mat[5] = arreglo_b[(tid * 16) + 10];
	mat[6] = arreglo_b[(tid * 16) + 12];
	mat[7] = arreglo_b[(tid * 16) + 13];
	mat[8] = arreglo_b[(tid * 16) + 14];
	
	det0 = mat[0]*(mat[4]*mat[8] - mat[5]*mat[7]);
	det1 = mat[1]*(mat[3]*mat[8] - mat[5]*mat[6]);
	det2 = mat[2]*(mat[3]*mat[7] - mat[4]*mat[6]);
	
	float result3 = det0 - det1 + det2;
	result3 *= arreglo_b[3];

	float result_total = result0 - result1 + result2 - result3;
	arreglo_a[tid] = result_total;
	
}

// realiza la suma de determinantes
__global__ void sumador_determinantes(int* arreglo, int* result, float N)
{
	__shared__ int compartida[threads_per_block];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	if(tid > N)
		return;
	
	compartida[threadIdx.x] = arreglo[tid];
	__syncthreads();
	for(int i=1; pow((float)2,(float)i-1) < threads_per_block; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1);
		if(threadIdx.x < ((float)threads_per_block/acceso) && (threadIdx.x * acceso + offset) < (N - blockIdx.x * blockDim.x))
		{
				compartida[threadIdx.x * acceso] = compartida[threadIdx.x * acceso] + compartida[threadIdx.x * acceso + offset];
				// compartida[threadIdx.x * acceso + offset] = 0;
		}
		__syncthreads();


	}

	//el primer thread de cada grupo guarda el resultado
	if(threadIdx.x == 0)
		result[blockIdx.x] = compartida[0];

}

__global__ void sumador_matrices(int* arreglo, int* result, float N)
{
	__shared__ int compartida[threads_per_block_matrix * 16];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid > N * 16)
		return;

	compartida[threadIdx.x] = arreglo[tid];
	__syncthreads();
	for(int i=1; pow((float)2,(float)i-1) < N; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1);

		int t_id = (threadIdx.x/16) * 16;
		int new_access = t_id * acceso + threadIdx.x % 16 ;
		int new_offset = new_access + offset * 16;

		if(t_id < ((float)threads_per_block_matrix*16/acceso) && (new_offset  < (threads_per_block_matrix*16)))
		{
				

				compartida[new_access] = compartida[new_access] + compartida[new_offset];
				// compartida[new_offset] = 0;
				// printf("GRUPO: %d - ITERACION: %d - TID %d - ACCESO: %d - OFFSET %d - REMAINING: %d \n", blockIdx.x,
				// 														i, 			tid, 	new_access , new_offset, threadIdx.x * acceso + offset);
		}
		__syncthreads();


	}

	//el primer thread de cada grupo guarda el resultado
	if(threadIdx.x < 16)
		result[blockIdx.x * 16 + threadIdx.x] = compartida[threadIdx.x];

}

int main(int argc, char** argv){

	int N = 4096*4096 ;

	int numBytesMatrices = sizeof(int) * N * 16; //bytes a alocar
	int numBytesDeterminantes = sizeof(int) * N; //bytes a alocar

	//##################################################################################
	//############################## INICIALIZACION ####################################

	suma_det = (int *) malloc(sizeof(int)); 

	arreglo_A = (int *) malloc(numBytesDeterminantes);	
	arreglo_B = (int *) malloc(numBytesDeterminantes);	


	mat_A = (int *) malloc(numBytesMatrices);
	mat_B = (int *) malloc(numBytesMatrices);

	arreglo_determinantes = (int*) malloc(numBytesDeterminantes);

	hipMalloc(&d_arreglo_determinantes, numBytesDeterminantes);
	hipMalloc(&d_arreglo_A, numBytesDeterminantes);
	hipMalloc(&d_arreglo_B, numBytesDeterminantes);

	hipMalloc(&d_mat_A, numBytesMatrices);
	hipMalloc(&d_mat_B, numBytesMatrices);

	init_CPU_matrices_array(mat_A, N);
	hipMemcpy(d_mat_A, mat_A, numBytesMatrices, hipMemcpyHostToDevice);


	//##################################################################################
	//################################ EJECUCIONES #####################################

	clock_t time_gpu = clock();

	//################################ DETERMINANTE ####################################
	dim3 miGrid1D_determinanteador(ceil((float)N/threads_per_block),1);
	dim3 miBloque1D_determinanteador(threads_per_block,1);

	determinanteador<<<miGrid1D_determinanteador,miBloque1D_determinanteador>>>(d_mat_A, d_arreglo_A, N);
	// hipDeviceSynchronize();
	// printf("ERROR %s\n", hipGetErrorString(hipGetLastError()));
	// hipMemcpy(arreglo_determinantes, d_arreglo_A, numBytesDeterminantes, hipMemcpyDeviceToHost);
	// print_CPU_matrix(arreglo_determinantes, N);



	//############################# SUMADOR DETERMINANTE ###############################

	dim3 miBloque1D_sumador(threads_per_block,1);
	for(int i=0; pow(threads_per_block, i) < N ; i++)
	{
		int remaining_elements = ceil((float)N/pow(threads_per_block, i));
		int block_count = ceil((float)N/pow(threads_per_block, i+1));
		dim3 miGrid1D_sumador(block_count,1);
		sumador_determinantes<<<miGrid1D_sumador, miBloque1D_sumador>>>(d_arreglo_A, d_arreglo_B, remaining_elements);
		hipDeviceSynchronize();
		// printf("ERROR: %s\n", hipGetErrorString(hipGetLastError()));
		// printf("elementos restantes: %d \n", remaining_elements);
		// printf("bloques usados: %d \n", block_count);
		int* tmp = d_arreglo_A;
		d_arreglo_A = d_arreglo_B;
		d_arreglo_B = tmp;
	}

	// hipMemcpy(arreglo_determinantes, d_arreglo_A, sizeof(int) * N, hipMemcpyDeviceToHost);
	// print_CPU_matrix(arreglo_determinantes, N);

	//############################## SUMADOR MATRICES ##################################

	dim3 miBloque1D_sumador_mat(threads_per_block,1);
	for(int i=0; pow(threads_per_block_matrix, i) < N ; i++)
	{
		int remaining_elements = ceil((float)N/pow(threads_per_block_matrix, i));
		int block_count = ceil((float)N/pow(threads_per_block_matrix, i+1));

		dim3 miGrid1D_sumador_mat(block_count,1);
		sumador_matrices<<<miGrid1D_sumador_mat, miBloque1D_sumador_mat>>>(d_mat_A, d_mat_B, remaining_elements);
		hipDeviceSynchronize();

		// printf("ERROR %s\n", hipGetErrorString(hipGetLastError()));
		// printf("elementos restantes: %d \n", remaining_elements);
		// printf("bloques usados: %d \n", block_count);
		int* tmp = d_mat_A;
		d_mat_A = d_mat_B;
		d_mat_B = tmp;

	}

	//############################### READ BACK ########################################

	clock_t time_gpu_end = clock();
	// PROMEDIO
	hipMemcpy(suma_det, d_arreglo_A, sizeof(int), hipMemcpyDeviceToHost);
	// printf("SUMA DE DETERMINANTES: %d\n", *suma_det);
	double promedio_det = (float)(*suma_det) / N;
	// printf("PROMEDIO: %lf\n", promedio_det);
	
	// SUMA DE MATRICES
	hipMemcpy(mat_B, d_mat_A, 16 * sizeof(int), hipMemcpyDeviceToHost);


	for(int i=0; i< 16; i++)
		mat_B[i] *= (int)promedio_det;

	printf("%s\n", "");

	printf("%s", "RESULTADO GPU:");
	print_CPU_matrix(mat_B, 16);



//        CCCCCCCCCCCCCPPPPPPPPPPPPPPPPP   UUUUUUUU     UUUUUUUU
//     CCC::::::::::::CP::::::::::::::::P  U::::::U     U::::::U
//   CC:::::::::::::::CP::::::PPPPPP:::::P U::::::U     U::::::U
//  C:::::CCCCCCCC::::CPP:::::P     P:::::PUU:::::U     U:::::UU
// C:::::C       CCCCCC  P::::P     P:::::P U:::::U     U:::::U 
//C:::::C                P::::P     P:::::P U:::::D     D:::::U 
//C:::::C                P::::PPPPPP:::::P  U:::::D     D:::::U 
//C:::::C                P:::::::::::::PP   U:::::D     D:::::U 
//C:::::C                P::::PPPPPPPPP     U:::::D     D:::::U 
//C:::::C                P::::P             U:::::D     D:::::U 
//C:::::C                P::::P             U:::::D     D:::::U 
// C:::::C       CCCCCC  P::::P             U::::::U   U::::::U 
//  C:::::CCCCCCCC::::CPP::::::PP           U:::::::UUU:::::::U 
//   CC:::::::::::::::CP::::::::P            UU:::::::::::::UU  
//     CCC::::::::::::CP::::::::P              UU:::::::::UU    
//        CCCCCCCCCCCCCPPPPPPPPPP                UUUUUUUUU      


	init_CPU_matrices_array(mat_A, N);
	for(int i=0; i< 16; i++)
	{
		mat_B[i] = 0;	
	}

	clock_t time_cpu = clock();
	for(int i=0; i< N; i++)
	{
		determinanteador_CPU(mat_A, arreglo_determinantes, i);
	}
	int suma_determinantes_cpu = 0;
	for(int i=0; i< N; i++)
	{
		suma_determinantes_cpu += arreglo_determinantes[i];
	}



	// printf("%s\n", "");
	// printf("%s\n", "");
	// printf("%s\n", "");
	// printf("%s\n", "DETERMINANTES CPU:");

	// print_CPU_matrix(arreglo_determinantes, N);
	// printf("%s\n", "");

	// printf("%s\n", "SUMA TOTAL DETERMINANTES:");
	// printf("%d\n", suma_determinantes_cpu);
	float promedio = (float)suma_determinantes_cpu / N;


	for(int j=0; j<16; j++)
	{
		for(int i=0; i< N; i++)
		{
			mat_B[j] += mat_A[i * 16 + j];
		}
		mat_B[j] *= promedio;
	}
	clock_t time_cpu_end = clock();


	printf("%s\n", "");
	printf("%s\n", "");
	printf("%s\n", "");
	printf("%s", "RESULTADO CPU:");

	print_CPU_matrix(mat_B, 16);


	printf("%s\n", "");
	printf("%s\n", "");
	printf("%s\n", "");

	printf("TIEMPO CPU: %lf\n", (double)(time_cpu_end - time_cpu) / CLOCKS_PER_SEC);
	printf("TIEMPO GPU: %lf\n", (double)(time_gpu_end - time_gpu) / CLOCKS_PER_SEC);



	free(arreglo_determinantes);
	free(suma_det);

	free(arreglo_A);
	free(arreglo_B);

	free(mat_A);
	free(mat_B);
	

	hipFree (d_arreglo_A);
	hipFree (d_arreglo_B);
	hipFree (d_arreglo_determinantes);

	hipFree (d_mat_A);
	hipFree (d_mat_B);


}

















