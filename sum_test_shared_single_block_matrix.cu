#include "hip/hip_runtime.h"
#include "stdio.h"
#define threads_per_block 10




void printi(int i){
	printf("%d\n", i);
}


void init_CPU_array(int* arreglo_b, int n){
	for(int i=0; i< n; i++)
	{
		arreglo_b[(i*16) + 0] = 1;
		arreglo_b[(i*16) + 1] = 1;
		arreglo_b[(i*16) + 2] = 1;
		arreglo_b[(i*16) + 3] = 1;
		arreglo_b[(i*16) + 4] = 1;
		arreglo_b[(i*16) + 5] = 1;
		arreglo_b[(i*16) + 6] = 1;
		arreglo_b[(i*16) + 7] = 1;
		arreglo_b[(i*16) + 8] = 1;
		arreglo_b[(i*16) + 9] = 1;
		arreglo_b[(i*16) + 10] = 1;
		arreglo_b[(i*16) + 11] = 1;
		arreglo_b[(i*16) + 12] = 1;
		arreglo_b[(i*16) + 13] = 1;
		arreglo_b[(i*16) + 14] = 1;
		arreglo_b[(i*16) + 15] = 1;

	}


}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}

void print_CPU_matrix(int array[], int n){
    for(int i = 0; i < n; i++) {
        if(i % 16 == 0)
            printf("%s\n", "");

        printf("%d ", array[i]);
    }
}


__global__ void sumador(int* arreglo, int* result, float N)
{
	__shared__ int compartida[threads_per_block * 16];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if(tid > N * 16)
		return;

	compartida[threadIdx.x] = arreglo[tid];
	__syncthreads();
	for(int i=1; pow((float)2,(float)i-1) < N; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1);

		int t_id = (threadIdx.x/16) * 16;
		int new_access = t_id * acceso + threadIdx.x % 16 ;
		int new_offset = t_id * acceso + offset * 16;

		if(t_id < (160.0/acceso) && (new_offset) < (N*16 - blockIdx.x * blockDim.x))
		{
				

				compartida[new_access] = compartida[new_access] + compartida[new_offset];
				compartida[new_offset] = 0;
				printf("TRABAJO ITERACION: %d - TID %d - ACCESO: %d - OFFSET %d - RESULTADO: %d \n", 
																		i, 			tid, 	t_id * acceso + threadIdx.x % 16 , t_id * acceso + offset * 16, compartida[t_id * acceso ]);
		}
		__syncthreads();


	}

	//el primer thread de cada grupo guarda el resultado
	if(threadIdx.x < 16)
		result[blockIdx.x * 16 + threadIdx.x] = compartida[threadIdx.x];

}





int* arreglo_suma1;
int* d_arreglo_suma1;

int* arreglo_result;
int* d_arreglo_suma2;

int main(int argc, char** argv){

	int N = 10;
	//##################################################################################
	//############################## INICIALIZACION ####################################
	int byte_size = N * sizeof(int) * 16;
	arreglo_suma1 = (int*) malloc(byte_size);
	hipMalloc(&d_arreglo_suma1, byte_size);

	arreglo_result = (int*) malloc(byte_size);
	hipMalloc(&d_arreglo_suma2, byte_size);


	init_CPU_array(arreglo_suma1, N);
	hipMemcpy(d_arreglo_suma1, arreglo_suma1, byte_size, hipMemcpyHostToDevice);


	//##################################################################################
	//################################ EJECUCIONES #####################################

	dim3 miBloque1D_1(threads_per_block * 16,1);
	dim3 miGrid1D_1(1,1);
	sumador<<<miGrid1D_1, miBloque1D_1>>>(d_arreglo_suma1, d_arreglo_suma2, N);

	//##################################################################################
	//################################### READ BACK #####################################

	hipMemcpy(arreglo_result, d_arreglo_suma2, N * sizeof(int) * 16, hipMemcpyDeviceToHost);

	printf("%s\n", "RESULTADO DE LA SUMA:");
	print_CPU_matrix(arreglo_result, N * 16);

	free(arreglo_suma1);
	hipFree (d_arreglo_suma1);

	free(arreglo_result);
	hipFree (d_arreglo_suma2);

}