#include "hip/hip_runtime.h"
#include "stdio.h"
#define threads_per_block 10




void printi(int i){
	printf("%d\n", i);
}


void init_CPU_array(int* arreglo_b, int n){
	for(int i=0; i< n; i++)
	{
		arreglo_b[(i*16) + 0] = 1;
		arreglo_b[(i*16) + 1] = 1;
		arreglo_b[(i*16) + 2] = 1;
		arreglo_b[(i*16) + 3] = 1;
		arreglo_b[(i*16) + 4] = 1;
		arreglo_b[(i*16) + 5] = 1;
		arreglo_b[(i*16) + 6] = 1;
		arreglo_b[(i*16) + 7] = 1;
		arreglo_b[(i*16) + 8] = 1;
		arreglo_b[(i*16) + 9] = 1;
		arreglo_b[(i*16) + 10] = 1;
		arreglo_b[(i*16) + 11] = 1;
		arreglo_b[(i*16) + 12] = 1;
		arreglo_b[(i*16) + 13] = 1;
		arreglo_b[(i*16) + 14] = 1;
		arreglo_b[(i*16) + 15] = 1;

	}


}
void print_CPU_array(int array[], int n){
	for(int i = 0; i < n; i++) {
		printi(array[i]);
	}
}


// realiza la suma de determinantes
__global__ void sumador(int* arreglo, int* result, float N)
{
	__shared__ int compartida[threads_per_block * 16];

	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	compartida[threadIdx.x] = arreglo[tid];
	printf("RESULT: %d, THREAD ID: %d\n", compartida[threadIdx.x], threadIdx.x);

	__syncthreads();
	for(int i=1; pow((float)2,(float)i-1) < 10 * 16; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1) * 16;
		if(threadIdx.x < ((10.0)/acceso) && (threadIdx.x * acceso + offset) < (N * 16- blockIdx.x * blockDim.x ))
		{
				compartida[threadIdx.x * acceso] = compartida[threadIdx.x * acceso] + compartida[threadIdx.x * acceso + offset];
				// compartida[threadIdx.x * acceso + offset * 16] = 0;

		}


	}

	//el primer thread de cada grupo guarda el resultado
	if(threadIdx.x < 16)
		result[blockIdx.x] = compartida[blockIdx.x];

}
//realiza la suma de n matrices
__global__ void sumador_2(int* arreglo, int* result, float N){

	__shared__ int compartido[threads_per_block * 16];
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	compartido[threadIdx.x] = arreglo[tid];
	__syncthreads();


	for(int i=1; pow((float)2,(float)i-1) < N; i++)
	{
		int acceso = pow((float)2,(float)i);
		int offset = pow((float)2, (float)i-1) ;
		if( (( (threadIdx.x / 16) & (offset*2)-1 ) == 0) && ( (threadIdx.x+offset*16) < N*16))
		{
				compartido[threadIdx.x] = compartido[threadIdx.x] + compartido[threadIdx.x + offset * 16];
				printf("TRABAJAMOS TID:%d - SUMA:%d + %d \n", threadIdx.x, threadIdx.x, threadIdx.x + offset * 16);
		}
	}
	__syncthreads();

	if(threadIdx.x < 16)
	{
		result[threadIdx.x] = compartido[threadIdx.x];
	}
}




int* arreglo_suma1;
int* d_arreglo_suma1;

int* arreglo_result;
int* d_arreglo_suma2;

int main(int argc, char** argv){

	int N = 4;
	//##################################################################################
	//############################## INICIALIZACION ####################################
	int byte_size = N * sizeof(int) * 16;
	arreglo_suma1 = (int*) malloc(byte_size);
	hipMalloc(&d_arreglo_suma1, byte_size);

	arreglo_result = (int*) malloc(byte_size);
	hipMalloc(&d_arreglo_suma2, byte_size);


	init_CPU_array(arreglo_suma1, N);
	hipMemcpy(d_arreglo_suma1, arreglo_suma1, byte_size, hipMemcpyHostToDevice);


	//##################################################################################
	//################################ EJECUCIONES #####################################

	dim3 miBloque1D_1(threads_per_block * 16,1);
	dim3 miGrid1D_1(1,1);
	sumador_2<<<miGrid1D_1, miBloque1D_1>>>(d_arreglo_suma1, d_arreglo_suma2, N);

	//##################################################################################
	//################################### READ BACK #####################################

	hipMemcpy(arreglo_result, d_arreglo_suma2, N * sizeof(int) * 16, hipMemcpyDeviceToHost);

	printf("%s\n", "RESULTADO DE LA SUMA:");
	print_CPU_array(arreglo_result, 16);

	free(arreglo_suma1);
	hipFree (d_arreglo_suma1);

	free(arreglo_result);
	hipFree (d_arreglo_suma2);

}